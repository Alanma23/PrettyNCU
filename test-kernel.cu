// Simple CUDA kernel with uncoalesced memory access pattern
// This kernel intentionally has poor memory access patterns for testing

#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 1024
#define STRIDE 128

// Kernel with uncoalesced memory access (performance issue)
__global__ void badMemoryAccess(float* input, float* output, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid < n) {
        // Uncoalesced access - threads access memory with large stride
        output[tid] = input[tid * STRIDE] * 2.0f;
    }
}

int main() {
    float *d_input, *d_output;
    size_t size = N * STRIDE * sizeof(float);

    // Allocate device memory
    hipMalloc(&d_input, size);
    hipMalloc(&d_output, N * sizeof(float));

    // Initialize input
    float* h_input = (float*)malloc(size);
    for (int i = 0; i < N * STRIDE; i++) {
        h_input[i] = (float)i;
    }
    hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice);

    // Launch kernel
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    badMemoryAccess<<<numBlocks, blockSize>>>(d_input, d_output, N);

    // Wait for completion
    hipDeviceSynchronize();

    // Check for errors
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        return 1;
    }

    // Cleanup
    hipFree(d_input);
    hipFree(d_output);
    free(h_input);

    printf("Kernel executed successfully\n");
    return 0;
}
